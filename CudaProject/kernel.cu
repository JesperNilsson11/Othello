#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstring>
#include <fstream>
#include "Window.h"
#include "console.h"
#include "Timer.h"

using namespace std;

std::ofstream of;
std::ofstream pos;
std::ofstream lines;
std::ofstream times;
std::ofstream nodes;

#define CUDA(call, string) cudaStatus = (call); if (cudaStatus != hipSuccess) {cout << string << " " << hipGetErrorString(cudaStatus) << endl; return cudaStatus;}
#define CUDAE(call) CUDA(call, "Error")
#define CUDANORETURN(call, string) cudaStatus = (call); if (cudaStatus != hipSuccess) {cout << string << " " << hipGetErrorString(cudaStatus) << endl; }
#define CHECK(call) CUDANORETURN(call, "Error")

hipError_t callCuda(int* data, int count);

int profileCopies(float        *h_a,
	float        *h_b,
	float        *d,
	unsigned int  n,
	char         *desc)
{
	hipError_t cudaStatus;
	cout << "\n" << desc <<  " transfers\n";

	unsigned int bytes = n * sizeof(float);

	// events for timing
	hipEvent_t startEvent, stopEvent;

	CUDAE(hipEventCreate(&startEvent));
	CUDAE(hipEventCreate(&stopEvent));

	CUDAE(hipEventRecord(startEvent, 0));
	CUDAE(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
	CUDAE(hipEventRecord(stopEvent, 0));
	CUDAE(hipEventSynchronize(stopEvent));

	float time;
	CUDAE(hipEventElapsedTime(&time, startEvent, stopEvent));
	cout << "  Host to Device bandwidth (GB/s): " << bytes * 1e-6 / time << " time: " << time << "\n";

	CUDAE(hipEventRecord(startEvent, 0));
	CUDAE(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
	CUDAE(hipEventRecord(stopEvent, 0));
	CUDAE(hipEventSynchronize(stopEvent));

	CUDAE(hipEventElapsedTime(&time, startEvent, stopEvent));
	cout << "  Device to Host bandwidth (GB/s): " << bytes * 1e-6 / time << "\n";

	for (int i = 0; i < n; ++i) {
		if (h_a[i] != h_b[i]) {
			cout << "*** " << desc << "transfers failed ***\n";
			break;
		}
	}

	// clean up events
	CUDAE(hipEventDestroy(startEvent));
	CUDAE(hipEventDestroy(stopEvent));

	return 0;
}

__device__ void partition(int* lp, int* rp, int pivot, int left, int right) {

}

__global__ void quicksort(int* data, int left, int right) {
	int nLeft, nRight;
	hipStream_t s1, s2;

	partition(data + left, data + right, data[left], nLeft, nRight);

	if (left < nRight) {
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort << < 1,1 >> > (data, left, nRight);
	}
	if (nLeft < right) {
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		quicksort << < 1,1 >> > (data, nLeft, right);
	}
}

__global__ void child() {
	printf("Hello\n");
}

__global__ void parent() {
	child << <1, 1 >> > ();
	child << <1, 1 >> > ();
	printf("World\n");
}

__device__ bool testMove(char* board, char x, char y, char dx, char dy, char player, char oppo) {
	x += dx;
	y += dy;
	
	if (x >= 0 && x < 8 && y >= 0 && y < 8) {
		if (board[y * 8 + x] != oppo) {
			return false;
		}
		else {
			x += dx;
			y += dy;
			while (x >= 0 && x < 8 && y >= 0 && y < 8) {
				if (board[y * 8 + x] == player) {
					return true;
				}
				else if (board[y * 8 + x] == ' ') {
					break;
				}
	
				x += dx;
				y += dy;
			}
		}
	}
	
	return false;
}

__device__ struct Move {
	bool dir[8];
	char move;
};

// NOT USED?
//__device__ bool possibleMove(char* board, char i, Move* move, char player) {
//	char x = i % 8;
//	char y = i / 8;
//	bool res = false;
//	char oppo = (player == 'P' ? 'O' : 'P');
//	if (board[x + y * 8] != ' ')
//		return false;
//
//	move->move = i;
//
//	move->dir[0] = testMove(board, x, y, -1, -1, player, oppo);
//	if (move->dir[0])
//		res = true;
//
//	move->dir[1] = testMove(board, x, y, 0, -1, player, oppo);
//	if (move->dir[1])
//		res = true;
//
//	move->dir[2] = testMove(board, x, y, 1, -1, player, oppo);
//	if (move->dir[2])
//		res = true;
//
//	move->dir[3] = testMove(board, x, y, -1, 0, player, oppo);
//	if (move->dir[3])
//		res = true;
//
//	move->dir[4] = testMove(board, x, y, 1, 0, player, oppo);
//	if (move->dir[4])
//		res = true;
//
//	move->dir[5] = testMove(board, x, y, -1, 1, player, oppo);
//	if (move->dir[5])
//		res = true;
//
//	move->dir[6] = testMove(board, x, y, 0, 1, player, oppo);
//	if (move->dir[6])
//		res = true;
//
//	move->dir[7] = testMove(board, x, y, 1, 1, player, oppo);
//	if (move->dir[7])
//		res = true;
//
//	return res;
//}

//__device__ bool possibleMove(char* board, char i, char* result, char player) {
//	char x = i % 8;
//	char y = i / 8;
//	bool res = false;
//	char oppo = (player == 'X' ? 'O' : 'X');
//	if (board[x + y * 8] != ' ')
//		return false;
//
//	result[0] = i;
//
//	result[1] = testMove(board, x, y, -1, -1, player, oppo);
//	if (result[1])
//		res = true;
//
//	result[2] = testMove(board, x, y, 0, -1, player, oppo);
//	if (result[2])
//		res = true;
//
//	result[3] = testMove(board, x, y, 1, -1, player, oppo);
//	if (result[3])
//		res = true;
//
//	result[4] = testMove(board, x, y, -1, 0, player, oppo);
//	if (result[4])
//		res = true;
//
//	result[5] = testMove(board, x, y, 1, 0, player, oppo);
//	if (result[5])
//		res = true;
//
//	result[6] = testMove(board, x, y, -1, 1, player, oppo);
//	if (result[6])
//		res = true;
//
//	result[7] = testMove(board, x, y, 0, 1, player, oppo);
//	if (result[7])
//		res = true;
//
//	result[8] = testMove(board, x, y, 1, 1, player, oppo);
//	if (result[8])
//		res = true;
//
//	return res;
//}

__device__ struct Moves {
	Move moves[20];
	int nr;
};

__global__ void updateMovesKernel(char* board, Moves* moves, char player) {
	__shared__ Move s_moves[64];
	//__shared__ char s_results[9 * 64];
	__shared__ bool s_bools[64];
	__shared__ char s_board[64];
	if (threadIdx.x == 0) {
		s_moves[threadIdx.y].move = threadIdx.y;
		s_bools[threadIdx.y] = false;
		//s_results[threadIdx.y * 9] = threadIdx.y;
		s_board[threadIdx.y] = board[threadIdx.y];
	}
	//if (blockIdx.x == 0) {
	//	//s_bools[0] = 1;
	//	s_results[0] = 1;
	//	s_results[1] = 2;
	//	s_results[2] = 3;
	//	s_results[3] = 4;
	//	s_results[4] = 5;
	//	s_results[5] = 6;
	//	s_results[6] = 7;
	//	s_results[9] = 8;
	//	//s_results[9+9] = 9;
	//}

	//if (threadIdx.x == 0 && threadIdx.y == 0)
	//	*(int*)results = 0;
	__syncthreads();// Not needed cuz sync further down?

	char x = threadIdx.y % 8;
	char y = threadIdx.y / 8;
	char dx;
	char dy;
	if (s_board[x + y * 8] == ' ') {
		if (threadIdx.x == 0 || threadIdx.x == 3 || threadIdx.x == 5)
			dx = -1;
		else if (threadIdx.x == 1 || threadIdx.x == 6)
			dx = 0;
		else
			dx = 1;

		if (threadIdx.x < 3)
			dy = -1;
		else if (threadIdx.x > 4)
			dy = 1;
		else
			dy = 0;


		//int index = threadIdx.y * 9 + threadIdx.x + 1;
		char oppo = 'O';
		if (player == 'O')
			oppo = 'P';
		s_moves[threadIdx.y].dir[threadIdx.x] = testMove(s_board, x, y, dx, dy, player, oppo);
		if (s_moves[threadIdx.y].dir[threadIdx.x]) {
			s_bools[threadIdx.y] = true;
		}
	}
	__syncthreads();

	if (threadIdx.x == 0) {
		//int id = atomicAdd((int*)(&moves->nr), 1);
		if (s_bools[threadIdx.y]) {
			int id = atomicAdd((int*)(&moves->nr), 1);

			//DEBUG CODE
			if (id >= 20)
				printf("To many moves!!!!!!!!!\n");
			//for (int i = 0; i < 9; ++i) {
			//	results[id * 9 + 4 + i] = s_results[threadIdx.y * 9 + i];
			//}

			moves->moves[id] = s_moves[threadIdx.y];
		}
	}
	//moves->nr = 11;
	// testing if working *(int*)results = 11;
}

//__global__ void updateMovesKernel(char* board, char* results, char player) {
//	__shared__ char s_results[9 * 64];
//	__shared__ char s_bools[64];
//	__shared__ char s_board[64];
//	if (threadIdx.x == 0) {
//		s_bools[threadIdx.y] = 0;
//		s_results[threadIdx.y * 9] = threadIdx.y;
//		s_board[threadIdx.y] = board[threadIdx.y];
//	}
//	if (blockIdx.x == 0) {
//		//s_bools[0] = 1;
//		s_results[0] = 1;
//		s_results[1] = 2;
//		s_results[2] = 3;
//		s_results[3] = 4;
//		s_results[4] = 5;
//		s_results[5] = 6;
//		s_results[6] = 7;
//		s_results[9] = 8;
//		//s_results[9+9] = 9;
//	}
//
//	if (threadIdx.x == 0 && threadIdx.y == 0)
//		*(int*)results = 0;
//	__syncthreads();// Not needed cuz sync further down?
//
//	char x = threadIdx.y % 8;
//	char y = threadIdx.y / 8;
//	char dx;
//	char dy;
//	if (s_board[x + y * 8] == ' ') {
//		if (threadIdx.x == 0 || threadIdx.x == 3 || threadIdx.x == 5)
//			dx = -1;
//		else if (threadIdx.x == 1 || threadIdx.x == 6)
//			dx = 0;
//		else
//			dx = 1;
//
//		if (threadIdx.x < 3)
//			dy = -1;
//		else if (threadIdx.x > 4)
//			dy = 1;
//		else
//			dy = 0;
//
//
//		int index = threadIdx.y * 9 + threadIdx.x + 1;
//		s_results[index] = testMove(s_board, x, y, dx, dy, player, (player == 'X' ? 'O' : 'X'));
//		if (s_results[index]) {
//			s_bools[threadIdx.y] = 1;
//		}
//	}
//	__syncthreads();
//
//	if (threadIdx.x == 0) {
//		if (s_bools[threadIdx.y]) {
//			int id = atomicAdd((int*)results, 1);
//			for (int i = 0; i < 9; ++i) {
//				results[id * 9 + 4 + i] = s_results[threadIdx.y * 9 + i];
//			}
//		}
//	}
//
//	// testing if working *(int*)results = 11;
//}

__device__ struct CUDANode {
	CUDANode* children[20];
	int score;
};

__device__ int calculateScore(char* b) {
	int P = 0;
	int O = 0;

	for (int x = 0; x < 64; ++x)
		if (b[x] == 'P')
			++P;
		else if (b[x] == 'O')
			++O;

	return P - O;
}

__device__ int d_write;

__device__ void moveDir(char* board, int x, int y, int dx, int dy, char player, int level) {
	int temp = x + y * 8;
	//char tempB[64];
	//for (int i = 0; i < 64; ++i)
	//	tempB[i] = board[i];
	x = x + dx;
	y = y + dy;

	while (board[y* 8 + x] != player) {
		board[y*8+x] = player;
		x += dx;
		y += dy;

		if (y < 0 || y > 7 || x < 0 || x > 7) {
			//int id = atomicAdd(&d_write, 1);
			//if (id == 0) {

			printf("Buggy cuda\nPLayer: %c\nlevel: %d index: %d dx: %d dy: %d\n", player, level, temp, dx, dy);
			//for (int i = 0; i < 64; ++i) {
			//	printf("%d%c ", i, tempB[i]);
			//	if (i % 8 == 7)
			//		printf("\n");
			//}
			//printf("===========\n");
			//for (int i = 0; i < 64; ++i) {
			//	printf("%d%c ", i, board[i]);
			//	if (i % 8 == 7)
			//		printf("\n");
			//}
			//d_write = 0;
			//}

			return;
		}
	}
}

__device__ void move(char* board, int i, char c, Move* move, int level) {
	int x = i % 8;
	int y = i / 8;

	//printf("move %d\n", i);
	if (move->dir[0])
		moveDir(board, x, y, -1, -1, c, level);
	if (move->dir[1])
		moveDir(board, x, y, 0, -1, c, level);
	if (move->dir[2])
		moveDir(board, x, y, 1, -1, c, level);

	if (move->dir[3])
		moveDir(board, x, y, -1, 0, c, level);
	if (move->dir[4])
		moveDir(board, x, y, 1, 0, c, level);

	if (move->dir[5])
		moveDir(board, x, y, -1, 1, c, level);
	if (move->dir[6])
		moveDir(board, x, y, 0, 1, c, level);
	if (move->dir[7])
		moveDir(board, x, y, 1, 1, c, level);
	
	//tempOrary bottom?
	board[i] = c;
}

__device__ char cudaBoard[64];
__device__ CUDANode headCuda;
__device__ int result;

__global__ void getMoveKernel(char* board, CUDANode* node, int level) {
	if (level == 5)
		d_write = 0;
	//printf("Start %d\n", level);
	if (level > 0) {
		char player = (level % 2 == 0) ? 'P' : 'O';
		
		Moves* moves = new Moves;
		moves->nr = 0;
		const dim3 threads(8, 64, 1);
		updateMovesKernel << <1, threads >> > (board, moves, player);
		hipDeviceSynchronize();
		char* newBoards[20];

		if (moves->nr == 0) {
			newBoards[0] = new char[64];
			memcpy(newBoards[0], board, 64);
			node->children[0] = new CUDANode;
			
			getMoveKernel << <1, 1 >> > (newBoards[0], node->children[0], level - 1);
			hipDeviceSynchronize();
			delete[] newBoards[0];
		}
		else {
			//printf("level %d\n", level);
			for (int i = 0; i < moves->nr; ++i) {
				newBoards[i] = new char[64];
				memcpy(newBoards[i], board, 64);
				node->children[i] = new CUDANode;
				move(newBoards[i], moves->moves[i].move, player, &moves->moves[i], level);


				getMoveKernel << <1, 1 >> > (newBoards[i], node->children[i], level - 1);
			}
			//printf("level %d\n", level);

			hipDeviceSynchronize();
			for (int i = 0; i < moves->nr; ++i)
				delete[] newBoards[i];
		}

		int temp = node->children[0]->score;
		int index = 0;
		if (player == 'P') {
			for (int i = 1; i < moves->nr; ++i) {
				if (temp < node->children[i]->score) {
					temp = node->children[i]->score;
					index = i;
				}
			}
		}
		else {
			for (int i = 1; i < moves->nr; ++i) {
				if (temp > node->children[i]->score) {
					temp = node->children[i]->score;
					index = i;
				}
				else if (temp == node->children[i]->score && moves->moves[i].move < moves->moves[index].move)
					index = i;// gpu chose lowest move same as cpu
			}
		}
		node->score = temp;

		if (level == 5) {
			//for (int i = 0; i < moves->nr; ++i)
			//	printf("%d ", moves->moves[i].move);
			result = moves->moves[index].move;
			printf("\nresult cuda %d\n", result);
		}
			

		//hipStream_t s1, s2;
		//node->left = new CUDANode;
		//hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		//getMoveKernel << <1, 1, 0,s1 >> > (node->left, level - 1);
		//hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		//node->right = new CUDANode;
		//getMoveKernel << <1, 1, 0,s2 >> > (node->right, level - 1);
		//hipDeviceSynchronize();
		//
		//node->score = node->left->score + node->right->score;

deletes:
		//delete node->left;
		//delete node->right;
		for (int i = 0; i < moves->nr; ++i)
			delete node->children[i];

		delete moves;
		//delete[] newBoard;
	}
	else {
		//printf("leaf node\n");
		//result = 11;
		node->score = calculateScore(board);
	}
	//printf("end %d\n", level);
	//char* board = new char[64];
	//for (int i = 0; i < 64; ++i)
	//	board[i] = ' ';
	//board[8 * 2 + 3] = 'P';
	//board[8 * 3 + 3] = 'P';
	//board[8 * 3 + 4] = 'P';
	//board[8 * 4 + 3] = 'P';
	//board[8 * 4 + 4] = 'O';
	//char* results = new char[4 + 9 * 10];
	//for (int i = 0; i < results[])
	//printf("number of moves: %d\n", (int)moves->nr);
	//for (int i = 0; i < moves->nr; ++i)
	//	printf("%d\n", moves->moves[i].move);
	//delete[] board;
	//delete[] results;
}

int WINAPI wWinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PWSTR lpCmdLine, int nCmdShow)
{
	RedirectIOToConsole();
	hipError_t cudaStatus;
	//hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 6);
	//CUDANode* cudaHead = nullptr;
	//CHECK(hipGetSymbolAddress((void**)&cudaHead, headCuda));
	//{
	//	Timer t;
	//	hipEvent_t startEvent, stopEvent;
	//	CHECK(hipEventCreate(&startEvent));
	//	CHECK(hipEventCreate(&stopEvent));
	//	CHECK(hipEventRecord(startEvent, 0));
	//	getMoveKernel << <1, 1 >> > (cudaHead, 5);
	//	CHECK(hipEventRecord(stopEvent, 0));
	//	CHECK(hipEventSynchronize(stopEvent));
	//
	//	float time;
	//	CHECK(hipEventElapsedTime(&time, startEvent, stopEvent));
	//	cout << "time: " << time << "ms\n";
	//	//hipDeviceSynchronize();
	//}
	//int score = 1;
	//void* symbol = nullptr;
	//cout << symbol;
	//CHECK(hipGetSymbolAddress(&symbol, HIP_SYMBOL(result)));
	//cout << " " << symbol << endl;
	//CHECK(hipMemcpy(&score, symbol, sizeof(int), hipMemcpyDeviceToHost));
	//cout << score << endl;
	//char* s = (char*)cudaHead;
	//s += (sizeof(CUDANode*) * 2);
	//CHECK(hipMemcpy(&score, (void*)s, sizeof(int), hipMemcpyDeviceToHost));
	//cout << score << endl;
	//
	//int temp;
	//cin >> temp;
	//
	//return score;
	//parent << <1, 1 >> > ();
	//CHECK(hipDeviceSynchronize());
	//cout << "hello" << endl;
	
	


	Window window(hInstance, nCmdShow);
	pos.open("data.txt");
	lines.open("lines.txt");
	times.open("times.txt");
	nodes.open("nodes.txt");
	of.open("console.txt", std::ios_base::app);
	of << "==============New Game==============" << std::endl;

	//hipError_t cudaStatus;
	cout << "start\n";
	{
		const unsigned int N = 1;//1048576;
	const unsigned int bytes = N * sizeof(int);
	int *h_a = new int[N];
	int *d_a;
	cout << "end" << endl;
	hipMalloc((int**)&d_a, bytes);

	memset(h_a, 0, bytes);
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	CUDAE(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
	hipFree(d_a);
	delete[] h_a;
	}

	//example 2
	unsigned int nElements = 1 * 1024 * 1024;
	const unsigned int bytes = nElements * sizeof(float);
	// host arrays
	float *h_aPageable, *h_bPageable;
	float *h_aPinned, *h_bPinned;
	// device array
	float *d_a;
	// allocate and initialize
	h_aPageable = new float[nElements];                    // host pageable
	h_bPageable = new float[nElements];                    // host pageable
	CUDAE(hipHostMalloc((void**)&h_aPinned, bytes)); // host pinned
	CUDAE(hipHostMalloc((void**)&h_bPinned, bytes)); // host pinned
	CUDAE(hipMalloc((void**)&d_a, bytes));           // device

	for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;

	memcpy(h_aPinned, h_aPageable, bytes);
	memset(h_bPageable, 0, bytes);
	memset(h_bPinned, 0, bytes);

	// output device info and transfer size
	hipDeviceProp_t prop;
	CUDAE(hipGetDeviceProperties(&prop, 0));

	cout << "\nDevice:" << prop.name << "\n";
	cout << "Transfer size (MB): " << bytes / (1024 * 1024) << "\n";

	// perform copies and report bandwidth
	profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
	profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");

	// cleanup
	hipFree(d_a);
	hipHostFree(h_aPinned);
	hipHostFree(h_bPinned);
	delete[] h_aPageable;
	delete[] h_bPageable;

	// call cuda function.
	int* data = nullptr;
	int count = 1024 * 1024;
	data = new int[count];
	cudaStatus = callCuda(data, count);
	if (cudaStatus != hipSuccess) {
		cout << "call to Cuda failed!";
		return 1;
	}
	delete[] data;


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		cout << "hipDeviceReset failed!";
		return 1;
	}

	//parent << <1, 1 >> > ();
	cout << "end" << endl;

	window.MSGLoop();
	of.close();
	pos.close();
	lines.close();
	times.close();
	nodes.close();

	return 0;
}

hipError_t callCuda(int* data, int count) {
	hipError_t cudaStatus;
	int* d_a;
	int bytes = count * 4;
	CUDAE(hipMalloc((void**)&d_a, bytes));
	CUDAE(hipMemcpy(d_a, data, bytes, hipMemcpyHostToDevice));
	//quicksort << <1,1 >> > (d_a, 0, count - 1);
	CUDAE(hipMemcpy(data, d_a, bytes, hipMemcpyDeviceToHost));
	hipFree(d_a);

	return hipSuccess;
}
char* getCurrentBoard();
int cudaGetMove() {
	hipError_t cudaStatus;
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 6);
	char* board = nullptr;
	CHECK(hipGetSymbolAddress((void**)&board, cudaBoard));
	CHECK(hipMemcpy(board, getCurrentBoard(), 64, hipMemcpyHostToDevice));

	CUDANode* cudaHead = nullptr;
	CHECK(hipGetSymbolAddress((void**)&cudaHead, headCuda));
	{
		Timer t;
		hipEvent_t startEvent, stopEvent;
		CHECK(hipEventCreate(&startEvent));
		CHECK(hipEventCreate(&stopEvent));
		CHECK(hipEventRecord(startEvent, 0));
		getMoveKernel << <1, 1 >> > (board, cudaHead, 5);
		CHECK(hipEventRecord(stopEvent, 0));
		CHECK(hipEventSynchronize(stopEvent));
		std::cout << "========================" << std::endl;

		float time;
		CHECK(hipEventElapsedTime(&time, startEvent, stopEvent));
		cout << "time: " << time << "ms\n";
		//hipDeviceSynchronize();
	}
	int score = 1;
	void* symbol = nullptr;
	cout << symbol;
	CHECK(hipGetSymbolAddress(&symbol, HIP_SYMBOL(result)));
	cout << " " << symbol << endl;
	CHECK(hipMemcpy(&score, symbol, sizeof(int), hipMemcpyDeviceToHost));
	cout << score << endl;
	char* s = (char*)cudaHead;
	s += (sizeof(CUDANode*) * 20);
	int nodeScore;
	CHECK(hipMemcpy(&nodeScore, (void*)s, sizeof(int), hipMemcpyDeviceToHost));
	cout << score << endl;

	return score;
}

/*
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
