#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstring>
#include <fstream>
#include "Window.h"

using namespace std;

std::ofstream of;
std::ofstream pos;
std::ofstream lines;

#define CUDA(call, string) cudaStatus = (call); if (cudaStatus != hipSuccess) {cout << string << " " << hipGetErrorString(cudaStatus) << endl; return cudaStatus;}
#define CUDAE(call) CUDA(call, "Error")

hipError_t callCuda(int* data, int count);

int profileCopies(float        *h_a,
	float        *h_b,
	float        *d,
	unsigned int  n,
	char         *desc)
{
	hipError_t cudaStatus;
	cout << "\n" << desc <<  " transfers\n";

	unsigned int bytes = n * sizeof(float);

	// events for timing
	hipEvent_t startEvent, stopEvent;

	CUDAE(hipEventCreate(&startEvent));
	CUDAE(hipEventCreate(&stopEvent));

	CUDAE(hipEventRecord(startEvent, 0));
	CUDAE(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
	CUDAE(hipEventRecord(stopEvent, 0));
	CUDAE(hipEventSynchronize(stopEvent));

	float time;
	CUDAE(hipEventElapsedTime(&time, startEvent, stopEvent));
	cout << "  Host to Device bandwidth (GB/s): " << bytes * 1e-6 / time << " time: " << time << "\n";

	CUDAE(hipEventRecord(startEvent, 0));
	CUDAE(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
	CUDAE(hipEventRecord(stopEvent, 0));
	CUDAE(hipEventSynchronize(stopEvent));

	CUDAE(hipEventElapsedTime(&time, startEvent, stopEvent));
	cout << "  Device to Host bandwidth (GB/s): " << bytes * 1e-6 / time << "\n";

	for (int i = 0; i < n; ++i) {
		if (h_a[i] != h_b[i]) {
			cout << "*** " << desc << "transfers failed ***\n";
			break;
		}
	}

	// clean up events
	CUDAE(hipEventDestroy(startEvent));
	CUDAE(hipEventDestroy(stopEvent));

	return 0;
}

__device__ void partition(int* lp, int* rp, int pivot, int left, int right) {

}

__global__ void quicksort(int* data, int left, int right) {
	int nLeft, nRight;
	hipStream_t s1, s2;

	partition(data + left, data + right, data[left], nLeft, nRight);

	if (left < nRight) {
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort << < 1,1 >> > (data, left, nRight);
	}
	if (nLeft < right) {
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		quicksort << < 1,1 >> > (data, nLeft, right);
	}
}

__global__ void child() {
	printf("Hello\n");
}

__global__ void parent() {
	child << <1, 1 >> > ();
	child << <1, 1 >> > ();
	printf("World\n");
}

int WINAPI wWinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PWSTR lpCmdLine, int nCmdShow)
{
	Window window(hInstance, nCmdShow);
	pos.open("data.txt");
	lines.open("lines.txt");
	of.open("console.txt", std::ios_base::app);
	of << "==============New Game==============" << std::endl;

	hipError_t cudaStatus;
	cout << "start\n";
	{
		const unsigned int N = 1;//1048576;
	const unsigned int bytes = N * sizeof(int);
	int *h_a = new int[N];
	int *d_a;
	cout << "end" << endl;
	hipMalloc((int**)&d_a, bytes);

	memset(h_a, 0, bytes);
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	CUDAE(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
	hipFree(d_a);
	delete[] h_a;
	}

	//example 2
	unsigned int nElements = 1 * 1024 * 1024;
	const unsigned int bytes = nElements * sizeof(float);
	// host arrays
	float *h_aPageable, *h_bPageable;
	float *h_aPinned, *h_bPinned;
	// device array
	float *d_a;
	// allocate and initialize
	h_aPageable = new float[nElements];                    // host pageable
	h_bPageable = new float[nElements];                    // host pageable
	CUDAE(hipHostMalloc((void**)&h_aPinned, bytes)); // host pinned
	CUDAE(hipHostMalloc((void**)&h_bPinned, bytes)); // host pinned
	CUDAE(hipMalloc((void**)&d_a, bytes));           // device

	for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;

	memcpy(h_aPinned, h_aPageable, bytes);
	memset(h_bPageable, 0, bytes);
	memset(h_bPinned, 0, bytes);

	// output device info and transfer size
	hipDeviceProp_t prop;
	CUDAE(hipGetDeviceProperties(&prop, 0));

	cout << "\nDevice:" << prop.name << "\n";
	cout << "Transfer size (MB): " << bytes / (1024 * 1024) << "\n";

	// perform copies and report bandwidth
	profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
	profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");

	// cleanup
	hipFree(d_a);
	hipHostFree(h_aPinned);
	hipHostFree(h_bPinned);
	delete[] h_aPageable;
	delete[] h_bPageable;

	// call cuda function.
	int* data = nullptr;
	int count = 1024 * 1024;
	data = new int[count];
	cudaStatus = callCuda(data, count);
	if (cudaStatus != hipSuccess) {
		cout << "call to Cuda failed!";
		return 1;
	}
	delete[] data;


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		cout << "hipDeviceReset failed!";
		return 1;
	}

	parent << <1, 1 >> > ();
	cout << "end" << endl;

	window.MSGLoop();
	of.close();
	pos.close();
	lines.close();

	return 0;
}

hipError_t callCuda(int* data, int count) {
	hipError_t cudaStatus;
	int* d_a;
	int bytes = count * 4;
	CUDAE(hipMalloc((void**)&d_a, bytes));
	CUDAE(hipMemcpy(d_a, data, bytes, hipMemcpyHostToDevice));
	quicksort << <1,1 >> > (d_a, 0, count - 1);
	CUDAE(hipMemcpy(data, d_a, bytes, hipMemcpyDeviceToHost));
	hipFree(d_a);

	return hipSuccess;
}

/*
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
